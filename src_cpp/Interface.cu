#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <fstream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>

struct Metadata
{
    int dim;
    int width;
    int height;
};

constexpr int getCountsSize(int binSize, int dim)
{
    return std::pow(binSize-1,dim);
}

template<typename inputDataType>
__device__ int getBinIndex(inputDataType value, const float* bins, int binCountPerDim) {
    for (int i = 0; i < binCountPerDim; ++i) {
        if (static_cast<float>(value) < bins[i + 1])
        {
            return i;
        }
    }
    return binCountPerDim-1;
}

template<typename inputDataType>
__global__ void histogramKernel(const inputDataType* inputImage, const Metadata metadata, const float* bins, int* counts, const int binCountPerDim) {
    int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    int tidY = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidX < metadata.width && tidY < metadata.height) {
        const inputDataType* pixel = &inputImage[tidY * metadata.width * metadata.dim + tidX * metadata.dim];
        int weight = 1;
        int countsIndex = 0;
        for(int dimIndex = metadata.dim-1; dimIndex >= 0; dimIndex--)
        {
            countsIndex += weight * getBinIndex<inputDataType>(pixel[dimIndex], &bins[dimIndex*(binCountPerDim+1)], binCountPerDim);
            weight *= binCountPerDim;
        }
        atomicAdd(&counts[countsIndex], 1);
    }
}

template<typename inputDataType>
void calculateHistogram(const inputDataType* hostImage, const Metadata metadata, const float* hostBins, int* hostCounts, int binSize) {
    inputDataType* deviceImage;
    float* deviceBins;
    int* deviceCounts;

    hipMalloc(&deviceImage, metadata.width * metadata.height * metadata.dim * sizeof(inputDataType));
    hipMalloc(&deviceBins, metadata.dim * binSize * sizeof(float));
    hipMalloc(&deviceCounts, getCountsSize(binSize, metadata.dim) * sizeof(int));

    hipMemcpy(deviceImage, hostImage, metadata.width * metadata.height * metadata.dim * sizeof(inputDataType), hipMemcpyHostToDevice);
    hipMemcpy(deviceBins, hostBins, metadata.dim * binSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(8,8);
    dim3 gridDim((metadata.width + blockDim.x - 1) / blockDim.x, (metadata.height + blockDim.y - 1) / blockDim.y);

    auto start = std::chrono::high_resolution_clock::now();
    histogramKernel<inputDataType><<<gridDim, blockDim>>>(deviceImage, metadata, deviceBins, deviceCounts, (binSize-1));
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time taken by histogram kernel: " << duration.count() << " microseconds" << std::endl;

    hipMemcpy(hostCounts, deviceCounts, getCountsSize(binSize, metadata.dim) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceImage);
    hipFree(deviceBins);
    hipFree(deviceCounts);
}


template<typename inputDataType>
__global__ void partialDecompositionKernel(const inputDataType* inputImage, const int row, const Metadata metadata, const float* bins, int* counts, const int binCountPerDim) {
    int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    const int imageRow = row * metadata.width * metadata.dim;
    const int countsRow = row * std::pow(binCountPerDim, metadata.dim);

    if (tidX < metadata.width) {
        const inputDataType* pixel = &inputImage[imageRow + tidX * metadata.dim];
        int weight = 1;
        int countsIndex = 0;
        for(int dimIndex = metadata.dim-1; dimIndex >= 0; dimIndex--)
        {
            countsIndex += weight * getBinIndex<inputDataType>(pixel[dimIndex], &bins[dimIndex*(binCountPerDim+1)], binCountPerDim);
            weight *= binCountPerDim;
        }
        atomicAdd(&counts[countsRow + countsIndex], 1);
    }
}

template<typename inputDataType>
void calculateRowPartialReduction(const inputDataType* hostImage, const Metadata metadata, const float* hostBins, int* hostCounts, int binSize) {
    inputDataType* deviceImage;
    float* deviceBins;
    int* deviceCounts;
    int numOfBlocksPerRow = std::ceil(metadata.width / 1024.0);
    int numOfThreadsPerBlock = metadata.width / numOfBlocksPerRow;

    hipMalloc(&deviceImage, metadata.width * metadata.height * metadata.dim * sizeof(inputDataType));
    hipMalloc(&deviceBins, metadata.dim * binSize * sizeof(float));
    hipMalloc(&deviceCounts, metadata.height * getCountsSize(binSize, metadata.dim) * sizeof(int));

    hipMemcpy(deviceImage, hostImage, metadata.width * metadata.height * metadata.dim * sizeof(inputDataType), hipMemcpyHostToDevice);
    hipMemcpy(deviceBins, hostBins, metadata.dim * binSize * sizeof(float), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();

    for(int row = 0; row < metadata.height; row++)
    {
        partialDecompositionKernel<inputDataType><<<numOfBlocksPerRow, numOfThreadsPerBlock>>>(deviceImage, row, metadata, deviceBins, deviceCounts, (binSize-1));
    }

    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time taken by partial reduction: " << duration.count() << " microseconds" << std::endl;

    hipMemcpy(hostCounts, deviceCounts, metadata.height * getCountsSize(binSize, metadata.dim) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceImage);
    hipFree(deviceBins);
    hipFree(deviceCounts);
}

int main(int argc, char** argv) {

    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <path_to_image>" << std::endl;
        return -1;
    }

    cv::Mat inputImage = cv::imread(argv[1]);
    if (inputImage.empty()) {
        std::cerr << "Error: Could not read the image." << std::endl;
        return -1;
    }

    Metadata metadata {.dim=3,
                       .width=inputImage.cols,
                       .height=inputImage.rows};
    std::cout << "Metadata height: " << metadata.height << " width: " << metadata.width << " dim: " << metadata.dim << std::endl;
    const int binSize = 5;
    using inputDataType = uint8_t;
    inputDataType* image = new inputDataType[metadata.width * metadata.height * metadata.dim];

    float bins[binSize * metadata.dim] = {0, 63.75, 127.5, 191.25, 255.0,
                                          0, 63.75, 127.5, 191.25, 255.0,
                                          0, 63.75, 127.5, 191.25, 255.0};
    int* counts = new int[getCountsSize(binSize, metadata.dim)];
    for (int y = 0; y < metadata.height; ++y) {
        for (int x = 0; x < metadata.width; ++x) {
            cv::Vec3b pixel = inputImage.at<cv::Vec3b>(y, x);

            // Assuming 1d array for image representation
            image[y * metadata.width * metadata.dim + x * metadata.dim + 0] = static_cast<inputDataType>(pixel[2]);
            image[y * metadata.width * metadata.dim + x * metadata.dim + 1] = static_cast<inputDataType>(pixel[1]);
            image[y * metadata.width * metadata.dim + x * metadata.dim + 2] = static_cast<inputDataType>(pixel[0]);
        }
    }

    calculateHistogram<inputDataType>(image, metadata, bins, counts, binSize);
    std::cout << "Histogram Counts:" << std::endl;
    for (int i = 0; i < getCountsSize(binSize, metadata.dim); ++i) {
        std::cout << counts[i] << " ";
        if ((i + 1) % (binSize-1) == 0)
        {
            std::cout << std::endl;
        }
    }
    std::ofstream myfile ("cudaCounts.txt");
    if (myfile.is_open())
    {
        for (int i = 0; i < getCountsSize(binSize, metadata.dim); ++i) {
            myfile << counts[i] << "\n";
        }
        myfile.close();
    }

    int* countsPartialReduction = new int[metadata.height * getCountsSize(binSize, metadata.dim)];
    calculateRowPartialReduction<inputDataType>(image, metadata, bins, countsPartialReduction, binSize);

    std::cout << "First row Counts:" << std::endl;
    int rowHistogramToPrint = 0;
    for (int i = 0; i < getCountsSize(binSize, metadata.dim); ++i) {
        std::cout << countsPartialReduction[rowHistogramToPrint * getCountsSize(binSize, metadata.dim) + i] << " ";
        if ((i + 1) % (binSize-1) == 0)
        {
            std::cout << std::endl;
        }
    }

    delete[] image;
    delete[] counts;
    delete[] countsPartialReduction;

    return 0;
}